#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>

/*
 *  Lectura Archivo
 */
void Read(float** R, float** G, float** B,int *L, int *M, int *N, const char *filename) {    
	FILE *fp;
	fp = fopen(filename, "r");
	fscanf(fp, "%d %d %d\n", L ,M, N); //ahora tambien leemos L

	//guardar todas los R,G,B de todas las imagenes en un solo array para R, G y B
	int imsize_total = (*L) * (*M) * (*N);
	int imsize = (*M) * (*N); //tamaño original de 1 imagen

	float* R1 = new float[imsize_total];
	float* G1 = new float[imsize_total];
	float* B1 = new float[imsize_total];

	for( int j=0; j < (*L); ++j){ //recorrer cantidad de imagenes

		//por cada bloque de imagenes asignar los indices
		for(int i = 0; i < imsize; i++)
			fscanf(fp, "%f ", &(R1[i + imsize*j]));
		for(int i = 0; i < imsize; i++)
			fscanf(fp, "%f ", &(G1[i + imsize*j]));
		for(int i = 0; i < imsize; i++)
			fscanf(fp, "%f ", &(B1[i + imsize*j]));
	} 
	fclose(fp);
	*R = R1; *G = G1; *B = B1;
}

/*
 *  Escritura Archivo
 */
void Write(float* R, float* G, float* B, int M, int N, const char *filename) {
	//no es neceario modificar nada, se asume R,G,B tamaño M*N con valores promediados
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", M, N);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", R[i]);
    fprintf(fp, "%f\n", R[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", G[i]);
    fprintf(fp, "%f\n", G[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", B[i]);
    fprintf(fp, "%f\n", B[M*N-1]);
    fclose(fp);
}

/*
 *  Procesamiento Imagen CPU
 */
void funcionCPU(float *R, float *G, float *B, float *Rout, float *Gout, float *Bout,int L, int M, int N){
	//con CPU iría dando saltos en los bloques de colores y sumando los valores y luego dividir por L
		int imsize = M*N;

		for( int i=0; i< imsize; ++i){ //por cada pixel en la imagen
			//calcular suma
			float sumaR = 0;
			float sumaG = 0;
			float sumaB = 0;
			for( int j=0; j < L; ++j){ //recorrer cantidad de imagenes
				sumaR += R[i + j*imsize];
				sumaG += G[i + j*imsize];
				sumaB += B[i + j*imsize];
			}
			Rout[i] = (sumaR)/L ;
			Gout[i] = (sumaG)/L ;
			Bout[i] = (sumaB)/L ;
		}
}

/*
 *  Procesamiento Imagen GPU
 */
__global__ void kernelGPU(float *R, float *G, float *B, float *Rout, float *Gout, float *Bout, int L,int M, int N){

}

/*
 *  Codigo Principal
 */
int main(int argc, char **argv){

    /*
     *  Inicializacion
     */
	int L, M, N;
    float *Rhost, *Ghost, *Bhost;
    float *Rhostout, *Ghostout, *Bhostout;
    float *Rdev, *Gdev, *Bdev;
    float *Rdevout, *Gdevout, *Bdevout;
    char names[2][3][30] = {
		//Nombres

		//-lee [0]------ escribe CPU [1]----- escribe GPU [2]
		{"test.txt\0", "testCPU.txt\0", "testGPU.txt\0"},
		{"images6.txt\0", "images6CPU.txt\0", "images6GPU.txt\0"},
		//{"imgG.txt\0", "imgGCPU.txt\0", "imgGGPU.txt\0"}
	
	};

    for (int i=0; i<2; i++){
	    Read(&Rhost, &Ghost, &Bhost, &L ,&M, &N, names[i][0]); // los ColorHost van a quedar de tamaño L*M*N

	    /*
	     *  CPU
	     */
		
		//los tamaños de estos no los modificamos porque vamos a hacer el promedio y quedan del tamaño original de 1 imagen
	    Rhostout = (float*)malloc(M*N*sizeof(float));
	    Ghostout = (float*)malloc(M*N*sizeof(float));
	    Bhostout = (float*)malloc(M*N*sizeof(float));

		/*
		Rhost -> L*M*N
		Ghost -> L*M*N
		Bhost -> L*M*N

		Rhostout -> M*N
		Ghostout -> M*N
		Bhostout -> M*N
		*/

	    funcionCPU(Rhost, Ghost, Bhost, Rhostout, Ghostout, Bhostout, L,M, N);  //pasarle L
	    Write(Rhostout, Ghostout, Bhostout, M, N, names[i][1]);

	    delete[] Rhostout; delete[] Ghostout; delete[] Bhostout;
	    



		//--------------------------------------- GPU -----------------------------------------------//
	    int grid_size, block_size = 256;
	    grid_size = (int)ceil((float) M * N / block_size);
		
		//es necesario ajustar los ColorDev (device) a tamaños L*M*N
		//pero los out no, quedan como M*N

		//reserva memoria
	    hipMalloc((void**)&Rdev, L * M * N * sizeof(float));
	    hipMalloc((void**)&Gdev, L * M * N * sizeof(float));
	    hipMalloc((void**)&Bdev, L * M * N * sizeof(float));

		//copia datos de ColorHost --> ColorDev
	    hipMemcpy(Rdev, Rhost, L * M * N * sizeof(float), hipMemcpyHostToDevice);
	    hipMemcpy(Gdev, Ghost, L * M * N * sizeof(float), hipMemcpyHostToDevice);
	    hipMemcpy(Bdev, Bhost, L * M * N * sizeof(float), hipMemcpyHostToDevice);
	        
	    hipMalloc((void**)&Rdevout, M * N * sizeof(float));
	    hipMalloc((void**)&Gdevout, M * N * sizeof(float));
	    hipMalloc((void**)&Bdevout, M * N * sizeof(float));
	    
		//GPU
	    kernelGPU<<<grid_size, block_size>>>(Rdev, Gdev, Bdev, Rdevout, Gdevout, Bdevout, L,M, N); //L

	    Rhostout = (float*)malloc(M*N*sizeof(float));
	    Ghostout = (float*)malloc(M*N*sizeof(float));
	    Bhostout = (float*)malloc(M*N*sizeof(float));
	    hipMemcpy(Rhostout, Rdevout, M * N * sizeof(float), hipMemcpyDeviceToHost);
	    hipMemcpy(Ghostout, Gdevout, M * N * sizeof(float), hipMemcpyDeviceToHost);
	    hipMemcpy(Bhostout, Bdevout, M * N * sizeof(float), hipMemcpyDeviceToHost);
	    Write(Rhostout, Ghostout, Bhostout, M, N, names[i][2]);

    	hipFree(Rdev); hipFree(Gdev); hipFree(Bdev);
    	hipFree(Rdevout); hipFree(Gdevout); hipFree(Bdevout);
    	free(Rhost); free(Ghost); free(Bhost);
    	free(Rhostout); free(Ghostout); free(Bhostout);
		//------------------------------------------------------------------------------------------//
	}
	return 0;
}